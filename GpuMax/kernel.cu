#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
int getSPcores(hipDeviceProp_t devProp)
{
	int cores = 0;
	int mp = devProp.multiProcessorCount;
	switch (devProp.major)
	{
	case 2: // Fermi
		if (devProp.minor == 1) cores = mp * 48;
		else cores = mp * 32;
		break;
	case 3: // Kepler
		cores = mp * 192;
		break;
	case 5: // Maxwell
		cores = mp * 128;
		break;
	case 6: // Pascal
		if (devProp.minor == 1) cores = mp * 128;
		else if (devProp.minor == 0) cores = mp * 64;
		else printf("Unknown device type\n");
		break;
	case 7: // Volta
		if (devProp.minor == 0) cores = mp * 64;
		else printf("Unknown device type\n");
		break;
	default:
		printf("Unknown device type\n");
		break;
	}
	return cores;
}
void print_cuda_device_info(hipDeviceProp_t& prop)
{
	printf("Device name:                                        %s\n", prop.name);
	printf("Global memory available on device:                  %zu\n", prop.totalGlobalMem);
	printf("Shared memory available per block:                  %zu\n", prop.sharedMemPerBlock);
	printf("Count of 32-bit registers available per block:      %i\n", prop.regsPerBlock);
	printf("Warp size in threads:                               %i\n", prop.warpSize);
	printf("Maximum pitch in bytes allowed by memory copies:    %zu\n", prop.memPitch);
	printf("Maximum number of threads per block:                %i\n", prop.maxThreadsPerBlock);
	printf("Maximum size of each dimension of a block[0]:       %i\n", prop.maxThreadsDim[0]);
	printf("Maximum size of each dimension of a block[1]:       %i\n", prop.maxThreadsDim[1]);
	printf("Maximum size of each dimension of a block[2]:       %i\n", prop.maxThreadsDim[2]);
	printf("Maximum size of each dimension of a grid[0]:        %i\n", prop.maxGridSize[0]);
	printf("Maximum size of each dimension of a grid[1]:        %i\n", prop.maxGridSize[1]);
	printf("Maximum size of each dimension of a grid[2]:        %i\n", prop.maxGridSize[2]);
	printf("Clock frequency in kilohertz:                       %i\n", prop.clockRate);
	printf("totalConstMem:                                      %zu\n", prop.totalConstMem);
	printf("Major compute capability:                           %i\n", prop.major);
	printf("Minor compute capability:                           %i\n", prop.minor);
	printf("Number of multiprocessors on device:                %i\n", prop.multiProcessorCount);
	printf("Count of cores:                                     %i\n", getSPcores(prop));

	// ...
}

	

__global__ void addVectors(float* a, float* b, float* c, int opPerThread) {
	int base = (blockIdx.x * blockDim.x + threadIdx.x) * opPerThread;
	for (int i = 0; i < opPerThread; ++i) {
		c[i + base] = a[i + base] + b[i + base];
	}
}

void dotest(int threadCount, int arrSize, int blockCount = 1024, int count = 1) {
	//const int arrSize = 10240000;
	//const int blockCount = 1024;
	//const int threadCount = 1000;
	int opPerThread = arrSize / (threadCount * blockCount);
	const int memSize = arrSize * sizeof(float);
	float* a = new float[arrSize];
	float* b = new float[arrSize];
	float* c = new float[arrSize];
	float time = clock();
	for (int i = 0; i < arrSize; ++i)
	{
		a[i] = i * 2.0f;
		b[i] = i * 0.5f;
	}
	hipError_t er = hipSetDevice(0);
	if (er != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	for (int i = 0; i < count; ++i) {
		float* dev_a;
		float* dev_b;
		float* dev_c;

		hipError_t
			err3 = hipMalloc(&dev_a, memSize);
		//if (hipError_t::hipSuccess != err3) {
		//	printf_s("Error in malloc dev_a #%i\n", err3);
		//}
		err3 = hipMalloc(&dev_b, memSize);
		/*if (hipError_t::hipSuccess != err3) {*/
			//	printf_s("Error in malloc dev_b #%i\n", err3);
			//}

		err3 = hipMalloc(&dev_c, memSize);
		//if (hipError_t::hipSuccess != err3) {
		//	printf_s("Error in malloc dev_с #%i\n", err3);
		//}
		hipError_t err1 = hipMemcpy(dev_a, a, memSize, hipMemcpyHostToDevice);;
		//if (hipError_t::hipSuccess != err1) {
		//	printf_s("Error in memcpy a to device #%i\n", err1);
		//}

		hipError_t err2 = hipMemcpy(dev_b, b, memSize, hipMemcpyHostToDevice);;
		//if (hipError_t::hipSuccess != err2) {
		//	printf_s("Error in memcpy b to device #%i\n", err2);
		//}

		float time = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);  //Записываем event

		addVectors << <blockCount, threadCount >> > (dev_a, dev_b, dev_c, opPerThread);

		hipError_t cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		//}
		//Создаем event
		cudaStatus = hipEventRecord(stop, 0);  //Записываем event
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipEventRecord returned error code %d\n", cudaStatus);

		//}
		cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

		//}
		cudaStatus = hipEventSynchronize(stop);
		/*if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipEventSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

		}*/
		//hipEventElapsedTime(&time, start, stop);
		//printf_s("threads %i blocks %i arrSize %i time %f", threadCount, blockCount, arrSize, time);
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, memSize, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed! %s\n", hipGetErrorString(cudaStatus));

		}//Синхронизируем event

		hipEventElapsedTime(&time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
		for (int i = 0; i < arrSize; ++i) {
			if (c[i] != i * 2 + i * 0.5f) {
				printf_s("Not match in c[%i] = %f in res = %f\n", i, c[i], i * 2 + i * 0.5f);
				break;
			};
		}
	}

	time -= clock();
	time = -time;
	time /= count;
	delete[] a;
	delete[] b;
	printf_s("Time of compute = %f milliseconds blockCount = %i threadCount = %i arrSize = %i\n", time, blockCount, threadCount, arrSize);

	delete[] c;

	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}

int main() {
	
	int threadCount = 1;
	int blockCount = 1;
	int arraySize = 10485760;
	//std::cout << "threadCount, blockCount, arraySize\n";
	//std::cin >> threadCount >> blockCount >> arraySize;
	///*while(threadCount <= 1024){
	//	dotest(threadCount, arraySize);
	//	threadCount *= 2;
	//}*/

	//dotest(threadCount, arraySize, blockCount, 1000);

	//printf_s("\n\n");

	//dotest(5, arraySize);
	////dotest(10, arraySize);
	//printf_s("\n\n");
	//arraySize = 10240000;
	//threadCount = 1;
	//while (threadCount <= 1024) {
	//	dotest(threadCount, arraySize);
	//	threadCount *= 10;
	//}
	//printf_s("\n\n");

	//dotest(8, arraySize);

	//dotest(20, arraySize);
	//dotest(40, arraySize);
	//dotest(80, arraySize);
	//dotest(1000, arraySize, 1000);

	//printf_s("\n\n\n\n\n");
	arraySize = 1000000;
	dotest(1, arraySize, 1000, 1000);
	dotest(1000, arraySize, 1, 1000);
	dotest(1000, arraySize, 1000, 1000);
	float* arr = new float[6]{ 1,2,3,4,5,6 };
	float time = clock();
	for (long long i = 0; i < 1000000000; ++i) {
		arr[i % 6] += arr[(i + 1) % 6];
	}
	time = clock() - time;
	delete[] arr;
	time /= 1000;
	printf_s("Time of compute = %f milliseconds on cpu\n", time);
	/*hipDeviceProp_t prop;
	cudaGetDeviceProperties_v2(&prop, 0);
	print_cuda_device_info(prop);*/
}