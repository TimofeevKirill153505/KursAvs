#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <cmath>

#define SIZE _rows * _columns * sizeof(float)

int Matrix::count = 0;

float prec1 = 1000000;

Matrix::Matrix(int rows, int columns) : _rows(rows), _columns(columns) {
	hipMalloc(&arr, _rows * _columns * sizeof(float));
}

__global__ void copyFunc(float* dst, float* src) {

}

Matrix::Matrix(const Matrix& other) :Matrix(other._rows, other._columns) {
	hipMemcpy(arr, other.arr, SIZE, hipMemcpyDeviceToDevice);
}

Matrix::Matrix(Matrix&& other) {
	arr = other.arr;
	_columns = other._columns;
	_rows = other._rows;
	other.arr = nullptr;
}

Matrix::Matrix() :_rows(0), _columns(0) {
}

Matrix& Matrix::operator=(const Matrix& other) {
	hipFree(arr);
	hipMalloc(&arr, SIZE);
	hipMemcpy(arr, other.arr, SIZE, hipMemcpyDeviceToDevice);
}

Matrix::~Matrix() {
	hipFree(arr);
}

__device__ void addMatrix(float* dst, float* src, int _rows, int _columns) {
	int i = blockIdx.x * _columns + threadIdx.x;
	dst[i] += src[i];
}

Matrix& Matrix::operator+=(const Matrix& other) {
	addMatrix <<<_rows, _columns >>> (arr, other._arr, _rows, _columns);
	hipDeviceSynchronize();
	return *this;
}

Matrix Matrix::operator+(const Matrix& other) {
	Matrix m (*this);

	return m += other;
}

__device__ void multOnFloat(float* src, float l, int _rows, int _columns) {
	src[blockIdx.x * _columns + threadIdx.x] *= l;
}

Matrix& Matrix::operator*=(float l) {
	multOnFloat<<<_rows, _columns>>>(arr, l, _rows, _columns);
	hipDeviceSynchronize();
	return* this;
}

Matrix Matrix::operator*(float l) {
	Matrix m(*this);

	return m *= l;
}

__device__ void divOnFloat(float* src, float l, int _rows, int _columns) {
	src[blockIdx.x * _columns + threadIdx.x] /= l;
}

Matrix& Matrix::operator/=(float l) {
	divOnFloat <<<_rows, _columns >>> (arr, l, _rows, _columns);
	hipDeviceSynchronize();
	return*this;
}

Matrix Matrix::operator/(float l) {
	Matrix m(*this);

	return m /= l;
}

Matrix& Matrix::operator=(Matrix&& other) {
	hipFree(arr);
	arr = other.arr;
	_columns = other._columns;
	_rows = other._rows;
	other.arr = nullptr;
}


__global__ void multRow(float* src, int row, float l, int rows, int columns) {
	int i = row * columns + threadIdx.x;
	src[i] *= l;
}

void Matrix::MultiplyRow(int row, float l) {
	multRow << <1, _columns >> > (arr, row, l,_rows, _columns);
	hipDeviceSynchronize();
}

__global__ void plusRows(float* src, int rowDst, int rowSrc, int rows, int columns) {
	int i = rowDst * columns + threadIdx.x;
	int i1 = rowSrc * columns + threadIdx.x;
	src[i] += src[i1];
}

void Matrix::PlusRows(int row1, int row2) {
	plusRows << <1, _columns >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void minusRows(float* src, int rowDst, int rowSrc, int rows, int columns) {
	int i = rowDst * columns + threadIdx.x;
	int i1 = rowSrc * columns + threadIdx.x;
	src[i] -= src[i1];
}

void Matrix::MinusRows(int row1, int row2) {
	minusRows << <1, _columns >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void swapRows(float* src, int row1, int row2, int rows, int columns) {
	int i = row1 * columns + threadIdx.x;
	int i1 = row2 * columns + threadIdx.x;
	float tmp = src[i];
	src[i] = src[i1];
	src[i1] = tmp;
}

void Matrix::swapLines(int line1, int line2) {
	swapRows << <1, _columns >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

void Matrix::ToUpTriangle() {
	int min = _rows < _columns ? _rows : _columns;

	for (int i = 0; i < min; ++i) {
		if (arr[i, i] == 0) break;

		int maxRow = i;
		for (int j = i + 1; j < _rows; ++j)
			if (arr[j, i] > arr[maxRow, i] && arr[j, i] != 0) maxRow = j;
		swapLines(i, maxRow);


		MultiplyRow(i, 1 / arr[i][i]);
		for (int j = i + 1; j < _rows; ++j) {
			float c = arr[j][i];
			if (c == 0) continue;

			MultiplyRow(i, c);
			MinusRows(j, i);
			MultiplyRow(i, 1 / c);

		}
	}
}

Matrix::operator std::string() const {
	std::string ans = "";

	for (int i = 0; i < _rows; i++) {
		for (int j = 0; j < _columns; ++j)
			ans += std::to_string((floorf(arr[i][j] * prec1) / prec1)) + " ";
		ans += "\n";
	}

	return ans;
}
