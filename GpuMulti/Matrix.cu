#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <cmath>

#define SIZE _rows * _columns * sizeof(float)

#define ERRORCHECKF(op) auto err = op; if(err != hipSuccess) std::cout << hipGetErrorString(err) << "\n";
#define ERRORCHECK(op) err = op; if(err != hipSuccess) std::cout << hipGetErrorString(err) << "\n";

int Matrix::count = 0;

float prec1 = 1000000;

__global__ void setZeros(float* arr, int _rows, int _columns) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	arr[i * _columns + j] = 0;
}

Matrix::Matrix(int rows, int columns) : _rows(rows), _columns(columns) {
	ERRORCHECKF( hipMalloc(&arr, _rows * _columns * sizeof(float)));
	setZeros << <_rows, _columns >> > (arr, _rows, _columns);
	hipDeviceSynchronize();
}


Matrix::Matrix(const Matrix& other) :Matrix(other._rows, other._columns) {
	//hipMalloc(&arr, SIZE);
	ERRORCHECKF(hipMemcpy(arr, other.arr, SIZE, hipMemcpyDeviceToDevice));
}

Matrix::Matrix(Matrix&& other) {
	arr = other.arr;
	_columns = other._columns;
	_rows = other._rows;
	other.arr = nullptr;
}

Matrix::Matrix() :_rows(0), _columns(0) {
}

Matrix& Matrix::operator=(const Matrix& other) {
	ERRORCHECKF(hipFree(arr));
	ERRORCHECK(hipMalloc(&arr, SIZE));
	ERRORCHECK(hipMemcpy(arr, other.arr, SIZE, hipMemcpyDeviceToDevice));
	return *this;
}

Matrix::~Matrix() {
	hipFree(arr);
}

__global__ void cudaGet(int i, int j, float* arr, float* ref, int _rows, int _columns) {
	//hipMemcpy(ref,&(arr[i * _columns + j]), sizeof(float), hipMemcpyDeviceToHost);
}

float Matrix::get(int i, int j) const{
	float ref = 0.1f;
	hipMemcpy(&ref, &(arr[i * _columns + j]), sizeof(float), hipMemcpyDeviceToHost);
	//cudaGet << <1, 1 >> > (i, j, arr, &ref, _rows, _columns);
	//ERRORCHECKF(hipDeviceSynchronize());
	return ref;
}

__global__ void cudaSet(int i, int j, float* arr, float value, int _rows, int _columns) {
	arr[i * _columns + j] = value;
}

void Matrix::set(int i, int j, float value) {
	cudaSet << <1, 1 >> > (i, j, arr, value, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void addMatrix(float* dst, float* src, int _rows, int _columns) {
	int i = blockIdx.x * _columns + threadIdx.x;
	dst[i] += src[i];
}

Matrix& Matrix::operator+=(const Matrix& other) {
	addMatrix <<<_rows, _columns >>> (arr, other.arr, _rows, _columns);
	hipDeviceSynchronize();
	return *this;
}

Matrix Matrix::operator+(const Matrix& other) {
	Matrix m (*this);

	return m += other;
}

__global__ void multOnFloat(float* src, float l, int _rows, int _columns) {
	src[blockIdx.x * _columns + threadIdx.x] *= l;
}

Matrix& Matrix::operator*=(float l) {
	multOnFloat<<<_rows, _columns>>>(arr, l, _rows, _columns);
	hipDeviceSynchronize();
	return* this;
}

Matrix Matrix::operator*(float l) {
	Matrix m(*this);

	return m *= l;
}

__global__ void divOnFloat(float* src, float l, int _rows, int _columns) {
	src[blockIdx.x * _columns + threadIdx.x] /= l;
}

Matrix& Matrix::operator/=(float l) {
	divOnFloat <<<_rows, _columns >>> (arr, l, _rows, _columns);
	hipDeviceSynchronize();
	return*this;
}

Matrix Matrix::operator/(float l) {
	Matrix m(*this);

	return m /= l;
}

Matrix& Matrix::operator=(Matrix&& other) {
	hipFree(arr);
	arr = other.arr;
	_columns = other._columns;
	_rows = other._rows;
	other.arr = nullptr;

	return *this;
}


__global__ void multRow(float* src, int row, float l, int rows, int columns) {
	int i = row * columns + threadIdx.x;
	src[i] *= l;
}

void Matrix::MultiplyRow(int row, float l) {
	multRow << <1, _columns >> > (arr, row, l,_rows, _columns);
	hipDeviceSynchronize();
}

__global__ void plusRows(float* src, int rowDst, int rowSrc, int rows, int columns) {
	int i = rowDst * columns + threadIdx.x;
	int i1 = rowSrc * columns + threadIdx.x;
	src[i] += src[i1];
}

void Matrix::PlusRows(int row1, int row2) {
	plusRows << <1, _columns >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void minusRows(float* src, int rowDst, int rowSrc, int rows, int columns) {
	int i = rowDst * columns + threadIdx.x;
	int i1 = rowSrc * columns + threadIdx.x;
	src[i] -= src[i1];
}

void Matrix::MinusRows(int row1, int row2) {
	minusRows << <1, _columns >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void swapRows(float* src, int row1, int row2, int rows, int columns) {
	int i = row1 * columns + threadIdx.x;
	int i1 = row2 * columns + threadIdx.x;
	float tmp = src[i];
	src[i] = src[i1];
	src[i1] = tmp;
}

void Matrix::swapLines(int line1, int line2) {
	swapRows << <6, _columns/6 >> > (arr, line1, line2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void currentColumnToZero(int current, float* arr, int _rows, int _columns) {
	int i = blockIdx.x + current + 1;
	int j = threadIdx.x + current;
	float f = arr[current * _columns + j];
	f *= arr[i * _columns + current];
	arr[i * _columns + j] -= f;
}

void Matrix::ToUpTriangle() {
	for (int i = 0; i < _rows; ++i) {
		MultiplyRow(i, 1 / get(i, i));
		if(i != _rows - 1) currentColumnToZero << <_rows - i - 1, _columns - i >> > (i, arr, _rows, _columns);
		hipDeviceSynchronize();
		//std::cout << "�� ������\n" << std::string(*this) << "\n\n";
	}
}

__global__ void backMoveFunc(float* x, float* arr, int current, int _rows, int _columns) {
	int i = threadIdx.x;
	x[i] += x[current] * arr[i * _columns + current];
}

float* Matrix::backMove(){
	float* x_d;
	hipMalloc(&x_d, sizeof(float) * _rows);
	setZeros << <1, _rows >> > (x_d, _rows, _columns);
	hipDeviceSynchronize();
	for (int i = _rows - 1; i >= 0; --i) {
		if (i == _rows - 1) {
			cudaSet << <1, 1 >> > (0, i, x_d, get(i, _columns - 1), _rows, _columns);
		}
		else {
			float sum = 0.1f;
			hipMemcpy(&sum, &(x_d[i]), sizeof(float), hipMemcpyDeviceToHost);

			//std::cout << i << " col - 1 " << get(i, _columns - 1) << "\n";
			//std::cout << i << " sum " << sum << "\n";

			cudaSet << <1, 1 >> > (0, i, x_d, get(i, _columns - 1) - sum, _rows, _columns);
			hipDeviceSynchronize();
		}

		if(i != 0 ) backMoveFunc << <1, i >> > (x_d, arr, i, _rows, _columns);
		hipDeviceSynchronize();
		//float* xux = new float[_rows];
		//hipMemcpy(xux, x_d, sizeof(float) * _rows, hipMemcpyDeviceToHost);
		//for(int g = 0; g < _rows; ++g) std::cout << "x" << g << " = " << xux[g] << " ";
		//std::cout << "\n";
		//delete[] xux;
	}
	float* x = new float[_rows];
	hipMemcpy(x, x_d, sizeof(float) * _rows, hipMemcpyDeviceToHost);
	return x;
}

Matrix::operator std::string() const {
	std::string ans = "";

	for (int i = 0; i < _rows; i++) {
		for (int j = 0; j < _columns; ++j)
			ans += std::to_string((floorf(get(i,j) * prec1) / prec1)) + " ";
		ans += "\n";
	}

	return ans;
}

__global__ void forMult(float* dst, float* arr1, float* arr2, int _rows, int _columns) {
	int  i = blockIdx.x;
	int j = threadIdx.x;
	dst[i * _columns + j] = arr1[i * _columns + i] * arr2[j * _columns + j];
}

Matrix Matrix::Multiply(const Matrix& other) {
	Matrix dst(_rows, _columns);
	forMult <<<_rows, _columns >>> (dst.arr, this->arr, other.arr, _rows, _columns);
	hipDeviceSynchronize();
	return dst;
}

__global__ void forDiffSquare(float* arr, int current, int _rows, int _columns) {
	int  i = blockIdx.x;
	int j = threadIdx.x;
	if(i != current && j != current)
	arr[i * _columns + j] =0;
}

Matrix Matrix::DiffSquare(int variable) {
	Matrix ret(*this);
	forDiffSquare<<<_rows,_columns>>>(ret.arr, variable, _rows, _columns);
	hipDeviceSynchronize();

	return ret;
}

__global__ void increm(float* arr, int i, int j, float inc, int _rows, int _columns) {
	arr[i * _columns + j] +=  inc;
}

void Matrix::Increase(int i, int j, float inc){
	increm<<<1, 1 >> > (arr, i, j, inc, _rows, _columns);
	hipDeviceSynchronize();

}
__global__ void copyDiffToMatrixFunc(float* arr, float* diffArr, int variable, int _rows, int _columns) {
	int j = threadIdx.x + 1;
	arr[(variable - 1) * _columns + j - 1] =
		diffArr[(variable) * _columns + j] + diffArr[(j) * _columns + variable];
}

void Matrix::CopyDiffToMatrix(Matrix& diff, int variable){
	copyDiffToMatrixFunc<<<1, _rows>>>(arr, diff.arr, variable, _rows, _columns);
	hipDeviceSynchronize();
	set(variable - 1, _columns - 1, -(diff.get(variable, 0) + diff.get(0, variable)));
}

__global__ void writeToDiagFunc(float* dst, float* src, int _rows, int _columns) {
	int i = threadIdx.x;
	
	dst[i * _columns + i] = src[i];
}

void Matrix::WriteToDiag(float* diagArr){
	float* src;
	hipMalloc(&src, _columns * sizeof(float));
	hipMemcpy(src, diagArr, _columns * sizeof(float), hipMemcpyHostToDevice);
	
	writeToDiagFunc << <1, _columns >> > (arr, src, _rows, _columns);
	hipDeviceSynchronize();;

	hipFree(src);
}