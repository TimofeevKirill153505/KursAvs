#include "hip/hip_runtime.h"
﻿#include <iostream>
#include "FuncRow.h"
#include "Matrix.h"
#include <functional>

const float PI = 3.1415926f;

typedef  std::function<Matrix(float x)> CoeffMatrixCounter;
typedef std::function<float(float)> Function;

//места для распараллеливания: гетУ вычисления матриц f(x)
//интеграл симпсона суммирование матриц
// в мульте возможно разбить на зоны
// в дифсквере сомнительно но можно попробовать

struct Piece {
	float a;
	float b;
};

struct Cond {
	float y1;
	float y2;
};

float prec2 = 10000000;

Matrix* GetY(CoeffMatrixCounter f, Piece piece, int numbOfPieces) {
	float h = (piece.b - piece.a) / numbOfPieces;

	float x = piece.a;
	Matrix* Y = new Matrix[numbOfPieces + 1];
	for (int i = 0; i < numbOfPieces + 1; ++i) {
		Y[i] = f(x);
		x += h;
		//std::cout << std::string(Y[i]) << "\n\n\n";
	}

	return Y;
}

Generator Anis(Cond cond, Piece piece) {
	return [cond, piece](int i)->Polynom {
		if (i == 0) {

			float k = (cond.y2 - cond.y1) / (piece.b - piece.a);
			//k*(x - piece.a) + cond.y1
			float c = cond.y1 - k * piece.a;
			float arr[] = { c,k,0 };
			//auto pol = Polynom(arr);
			//std::cout << std::string(pol) << "\n";
			return Polynom(arr);
		}

		float arr2[3] = { piece.a * piece.b, -(piece.a + piece.b), 1 };
		Polynom polly(arr2);
		//Polynom pol(i - 1);
		//pol[i - 1] = 1;
		//std::cout << (std::string)(pol * polly) << "\n";
		return polly.IncreasePow(i - 1);
		};
}

Matrix IntSimpson(CoeffMatrixCounter f, int numbOfPieces, Piece piece) {
	float h = (piece.b - piece.a) / numbOfPieces;
	//Matrix* Y = GetY(f, piece, numbOfPieces);
	float x = piece.a;
	//Matrix* Y = new Matrix[numbOfPieces + 1];
	Matrix s = f(x) + f(piece.b);
	x += h;
	//std::cout << std::string(s) << "\n\n";
	for (int i = 1; i < numbOfPieces; ++i) {
		if (i % 2 == 0) s += f(x) * 2;
		else s += f(x) * 4;
		x += h;
		
	}

	return s * (h / 3);
}

//Matrix IntSimpson(CoeffMatrixCounter f, int numbOfPieces, Piece piece) {
//	Matrix* Y = GetY(f, piece,
//		numbOfPieces); //должен возвращать матрицы квадрата невязки (возвращает то что возвращает и не ебет)
//	float h = (piece.b - piece.a) / numbOfPieces;
//	Matrix s = Y[0] + Y[numbOfPieces];
//	for (int i = 1; i < numbOfPieces; ++i)
//		if (i % 2 == 0) s += Y[i] * 2;
//		else s += Y[i] * 4;
//
//	delete[] Y;
//	return s * h / 3;
//}

Matrix Mult(Matrix& m1, Matrix& m2) {
	return m1.Multiply(m2);
}

Matrix DiffSquare(Matrix& m, int variable) {
	return m.DiffSquare(variable);
}

FuncRow MnkInt(Function p, Function q, Function f, int numbOfMembers, Generator memb, int numbOfPieces,
	Piece piece) {
	FuncRow frow(numbOfMembers, memb);
	//std::cout << std::string(frow) << "\n\n\n";

	auto lambda =
		[numbOfMembers, frow, p, q, f](float x)->Matrix {
		static int counter = 0;
		Matrix A(numbOfMembers, numbOfMembers);
		//std::cout << std::string(frow) << "\n\n";
		float* incArr = new float[numbOfMembers];
		for (int i = 0; i < numbOfMembers; ++i) {
			incArr[i] = frow[i].Diff().Diff().Count(x);
			//A.Increase(i,i, frow[i].Diff().Diff().Count(x));
			//std::cout << i << "th number " << A.get(i,i) <<  " delta " << frow[i].Diff().Diff().Count(x) <<" ";
			incArr[i] += frow[i].Diff().Count(x) * p(x);
			//std::cout << A.get(i, i) << " delta " << frow[i].Diff().Count(x) * p(x) << " ";
			incArr[i] += frow[i].Count(x) * q(x);
			//std::cout << A.get(i, i) <<  " delta " << frow[i].Count(x) * q(x) << "\n";

		}

		incArr[0] -= f(x);
		A.WriteToDiag(incArr);
		delete[] incArr;
		auto M = Mult(A, A);
		//std::cout << "A" << counter <<" = " << "\n";
		//std::cout << std::string(M) << "\n\n\n";
		++counter;
		return M;
		};

	Matrix intgr = IntSimpson(lambda, numbOfPieces, piece);
	//std::cout << std::string(intgr) << "\n\n\n";

	Matrix matr(numbOfMembers - 1, numbOfMembers);

	for (int i = 1; i < numbOfMembers; ++i) {
		Matrix df = DiffSquare(intgr, i);

		matr.CopyDiffToMatrix(df, i);
	}

	//std::cout << "До треуголирования\n" << std::string(matr) << "\n\n";
	matr.ToUpTriangle();
	//std::cout << "После треуголирования\n" << std::string(matr) << "\n\n";
	float* coeffs = matr.backMove();

	for (int i = 1; i < numbOfMembers; ++i) frow[i] *= coeffs[i - 1];

	delete[] coeffs;

	return frow;
}

void ShowDataTest(FuncRow frow, Function ans, Piece piece, int numbOfPoints) {
	//std::cout << ("Полученный ряд:\n");
	//std::cout << (std::string)frow;

	float h = (piece.b - piece.a) / numbOfPoints;
	float x = piece.a;
	float* delt = new float[numbOfPoints + 1];
	for (int i = 0; i <= numbOfPoints; ++i, x += h) {
		delt[i] = abs(frow.Count(x) - ans(x));
		//std::cout << myround(x, prec2) <<
		//	" Полученное решение : " << myround(frow.Count(x), prec2) <<
		//	"   Ответ: " << myround(ans(x), prec2) << "  Невязка: " <<
		//	myround(delt[i], prec2) << "\n";
	}

	float Norm = 0;
	for (int i = 0; i < numbOfPoints + 1; ++i) if (Norm < delt[i]) Norm = delt[i];
	delete[] delt;
	std::cout << "Норма невязки: " << Norm << "\n";
}

void dotest(Function p_t, Function q_t, Function f_t, Function lambda, Piece piece,
	Cond cond, int i, int numbOfMembers, int numbOfPoints) {
	std::cout << "Интегральный МНК Тестовый пример " << i << "\n";
	auto beg = clock();
	FuncRow frow = MnkInt(p_t, q_t, f_t, numbOfMembers, Anis(cond, piece), 30, piece);
	auto end = clock();
	std::cout << "Время выполнения " << end - beg << " миллисекунд" << "\n";
	ShowDataTest(frow, lambda, piece, numbOfPoints);
}

int main() {

	setlocale(0, "");
	Matrix A(5, 5);
	A.Increase(1, 1, -2);
	std::cout << std::string(A) << "\n\n";
	A.Increase(2, 2, 2);
	int i = 1;
	Function p_t = [](float x)->float { return 0; };
	Function q_t = [](float x) { return 1; };
	Function f_t = [](float x)->float { return x * x + 3 * x - 7; };
	Function lambda = [](float x)->float { return x * x + 3 * x - 9; };

	Piece piece_t = { 0, 1 };
	Cond cond_t = { -9, -5 };
	const int numbOfMembers = 1000;
	const int numbOfPoints = 50;
	//Function lambda = (x) => { return 0; };

	/*for (int i = 1; i <= 4; ++i) {
		Polynom pol (i + 2);
		pol[i + 2] = -1;
		pol[i] = 1;
		std::cout << std::string(Anis(cond_t, piece_t)(i)) << " и " << std::string(pol) << std::endl;
	}*/

	dotest(p_t, q_t, f_t, lambda, piece_t, cond_t, i, numbOfMembers, numbOfPoints);

	Function e3 = [](float x)->float {return expf(3 * x); };
	p_t = [](float x)->float {return  x * x + 6; };
	q_t = [](float x)->float {return  pow(2, -x); };
	f_t = [e3](float x)->float {return  9 * e3(x) + 3 * (x * x + 6) * e3(x) + pow(2, -x) * e3(x); };
	lambda = [](float x)->float {return  expf(3 * x); };
	piece_t = { 0, 1 };
	cond_t = { 1, expf(3) };
	i = 2;

	dotest(p_t, q_t, f_t, lambda, piece_t, cond_t, i, numbOfMembers, numbOfPoints);

	p_t = [](float x)->float {return  0; };
	q_t = [](float x)->float {return 1; };
	f_t = [](float x)->float {return  2 * x - PI; };
	lambda = [](float x)->float {return 2 * x - PI + PI * cos(x); };
	cond_t = { 0, lambda(1) };
	piece_t = { 0, 1 };

	i = 3;
	dotest(p_t, q_t, f_t, lambda, piece_t, cond_t, i, numbOfMembers, numbOfPoints);


	p_t = [](float x)->float {return  2; };
	q_t = [](float x)->float {return  1; };
	f_t = [](float x)->float {return  0; };
	lambda = [](float x)->float {return  expf(-x) + x * expf(-x); };
	piece_t = { 0, 1 };
	cond_t = { lambda(0), lambda(1) };

	i = 4;
	dotest(p_t, q_t, f_t, lambda, piece_t, cond_t, i, numbOfMembers, numbOfPoints);

}