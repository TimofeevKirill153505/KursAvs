#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <cmath>

#define SIZE _rows * _columns * sizeof(float)

#define ERRORCHECKF(op) auto err = op; if(err != hipSuccess) std::cout << hipGetErrorString(err) << "\n";
#define ERRORCHECK(op) err = op; if(err != hipSuccess) std::cout << hipGetErrorString(err) << "\n";

int Matrix::count = 0;

float prec1 = 1000000;

__global__ void setZeros(float* arr, int _rows, int _columns) {
	for (int i = 0; i < _rows; ++i)
		for (int j = 0; j < _columns; ++j)
			arr[i * _columns + j] = 0;
}

Matrix::Matrix(int rows, int columns) : _rows(rows), _columns(columns) {
	ERRORCHECKF(hipMalloc(&arr, _rows * _columns * sizeof(float)));
	setZeros << <1, 1 >> > (arr, _rows, _columns);
	hipDeviceSynchronize();
}


Matrix::Matrix(const Matrix& other) :Matrix(other._rows, other._columns) {
	//hipMalloc(&arr, SIZE);
	ERRORCHECKF(hipMemcpy(arr, other.arr, SIZE, hipMemcpyDeviceToDevice));
}

Matrix::Matrix(Matrix&& other) {
	arr = other.arr;
	_columns = other._columns;
	_rows = other._rows;
	other.arr = nullptr;
}

Matrix::Matrix() :_rows(0), _columns(0) {
}

Matrix& Matrix::operator=(const Matrix& other) {
	ERRORCHECKF(hipFree(arr));
	ERRORCHECK(hipMalloc(&arr, SIZE));
	ERRORCHECK(hipMemcpy(arr, other.arr, SIZE, hipMemcpyDeviceToDevice));


	return *this;
}

Matrix::~Matrix() {
	hipFree(arr);
}

__global__ void cudaGet(int i, int j, float* arr, float* ref, int _rows, int _columns) {
	//hipMemcpy(ref,&(arr[i * _columns + j]), sizeof(float), hipMemcpyDeviceToHost);
}

float Matrix::get(int i, int j) const {
	float ref = 0.1f;
	hipMemcpy(&ref, &(arr[i * _columns + j]), sizeof(float), hipMemcpyDeviceToHost);
	//cudaGet << <1, 1 >> > (i, j, arr, &ref, _rows, _columns);
	//ERRORCHECKF(hipDeviceSynchronize());
	return ref;
}

__global__ void cudaSet(int i, int j, float* arr, float value, int _rows, int _columns) {
	arr[i * _columns + j] = value;
}

void Matrix::set(int i, int j, float value) {
	cudaSet << <1, 1 >> > (i, j, arr, value, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void addMatrix(float* dst, float* src, int _rows, int _columns) {
	for (int i = 0; i < _rows * _columns; ++i)
		dst[i] += src[i];
}

Matrix& Matrix::operator+=(const Matrix& other) {
	addMatrix << <1, 1 >> > (arr, other.arr, _rows, _columns);
	hipDeviceSynchronize();
	return *this;
}

Matrix Matrix::operator+(const Matrix& other) {
	Matrix m(*this);

	return m += other;
}

__global__ void multOnFloat(float* src, float l, int _rows, int _columns) {
	for (int i = 0; i < _rows * _columns; ++i)
		src[i] *= l;
}

Matrix& Matrix::operator*=(float l) {
	multOnFloat << <1, 1 >> > (arr, l, _rows, _columns);
	hipDeviceSynchronize();
	return*this;
}

Matrix Matrix::operator*(float l) {
	Matrix m(*this);

	return m *= l;
}

__global__ void divOnFloat(float* src, float l, int _rows, int _columns) {
	for (int i = 0; i < _rows * _columns; ++i)
		src[i] /= l;
}

Matrix& Matrix::operator/=(float l) {
	divOnFloat << <1, 1 >> > (arr, l, _rows, _columns);
	hipDeviceSynchronize();
	return*this;
}

Matrix Matrix::operator/(float l) {
	Matrix m(*this);

	return m /= l;
}

Matrix& Matrix::operator=(Matrix&& other) {
	hipFree(arr);
	arr = other.arr;
	_columns = other._columns;
	_rows = other._rows;
	other.arr = nullptr;
	return *this;
}


__global__ void multRow(float* src, int row, float l, int rows, int columns) {
	for (int i = columns * (row); i < (row + 1) * columns; ++i)
		src[i] *= l;

}

void Matrix::MultiplyRow(int row, float l) {
	//std::cout << "columns " << _columns << " blocks " << _blocksForRow << " threads " << _threadsForRow << "\n";
	multRow << <1, 1 >> > (arr, row, l, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void plusRows(float* src, int rowDst, int rowSrc, int rows, int columns) {
	for (int i = (rowDst - 1) * columns, i1 = (rowSrc - 1) * columns; i < rowDst * columns; ++i, ++i1)
		src[i] += src[i1];
}

void Matrix::PlusRows(int row1, int row2) {
	plusRows << <1, 1 >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void minusRows(float* src, int rowDst, int rowSrc, int rows, int columns) {
	for (int i = (rowDst) * columns, i1 = rowSrc * columns; i < (rowDst + 1) * columns; ++i, ++i1)
		src[i] -= src[i1];
}

void Matrix::MinusRows(int row1, int row2) {
	minusRows << <1, 1 >> > (arr, row1, row2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void swapRows(float* src, int row1, int row2, int rows, int columns) {
	for (int i = (row1 - 1) * columns, i1 = (row2 - 1) * columns; i < row1 * columns; ++i, ++i1) {
		float tmp = src[i];
		src[i] = src[i1];
		src[i1] = tmp;
	}
}

void Matrix::swapLines(int line1, int line2) {
	swapRows << <1, 1 >> > (arr, line1, line2, _rows, _columns);
	hipDeviceSynchronize();
}

__global__ void currentColumnToZero(int current, float* arr, int _rows, int _columns) {	
	for (int i = current + 1; i < _rows; ++i) {
		float c = arr[i * _columns + current];
		for (int j = current; j < _columns; ++j) {
			float f = arr[current * _columns + j];
			f *= c;
			arr[i * _columns + j] -= f;
		}
	}
}

void Matrix::ToUpTriangle() {
	for (int i = 0; i < _rows; ++i) {
		float ii = get(i, i);
		if (ii == 0) continue;
		MultiplyRow(i, 1 / get(i, i));

		//std::cout << "�� ������ ��\n";// << std::string(*this) << "\n\n";
		if (i != _rows - 1) currentColumnToZero << <1, 1 >> > (i, arr, _rows, _columns);
		hipDeviceSynchronize();
		//std::cout << "�� ������ �����\n" << std::string(*this) << "\n\n";
		ERRORCHECKF(hipGetLastError());
	}
	//std::cout << "blocks " << _blocksForRow << " threads " << _threadsForRow << " columns " << _columns << "\n";
}

__global__ void backMoveFunc(float* x, float* arr, int current, int _rows, int _columns) {
	for (int i = 0; i < current; ++i) {
		x[i] += x[current] * arr[i * _columns + current];
	}
}

float* Matrix::backMove() {
	float* x_d;
	hipMalloc(&x_d, sizeof(float) * _rows);
	setZeros << <1, 1 >> > (x_d, 1, _rows);
	hipDeviceSynchronize();
	for (int i = _rows - 1; i >= 0; --i) {
		if (i == _rows - 1) {
			cudaSet << <1, 1 >> > (0, i, x_d, get(i, _columns - 1), _rows, _columns);
		}
		else {
			float sum = 0.1f;
			hipMemcpy(&sum, &(x_d[i]), sizeof(float), hipMemcpyDeviceToHost);

			//std::cout << i << " col - 1 " << get(i, _columns - 1) << "\n";
			//std::cout << i << " sum " << sum << "\n";

			cudaSet << <1, 1 >> > (0, i, x_d, get(i, _columns - 1) - sum, _rows, _columns);
			hipDeviceSynchronize();
		}

		if (i != 0) {
			backMoveFunc << <1, 1 >> > (x_d, arr, i, _rows, _columns);
			hipDeviceSynchronize();
		}
		//float* xux = new float[_rows];
		//hipMemcpy(xux, x_d, sizeof(float) * _rows, hipMemcpyDeviceToHost);
		//for(int g = 0; g < _rows; ++g) std::cout << "x" << g << " = " << xux[g] << " ";
		//std::cout << "\n";
		//delete[] xux;
	}
	float* x = new float[_rows];
	hipMemcpy(x, x_d, sizeof(float) * _rows, hipMemcpyDeviceToHost);
	return x;
}

Matrix::operator std::string() const {
	std::string ans = "";

	for (int i = 0; i < _rows; i++) {
		for (int j = 0; j < _columns; ++j)
			ans += std::to_string(get(i, j)) + " ";
		ans += "\n";
	}

	return ans;
}

__global__ void forMult(float* dst, float* arr1, float* arr2, int _rows, int _columns) {
	for (int i = 0; i < _rows; ++i) {
		for (int j = 0; j < _columns; ++j) {
			dst[i * _columns + j] = arr1[i * _columns + i] * arr2[j * _columns + j];
		}
	}
}

Matrix Matrix::Multiply(const Matrix& other) {
	Matrix dst(_rows, _columns);
	forMult << <1, 1 >> > (dst.arr, this->arr, other.arr, _rows, _columns);
	hipDeviceSynchronize();
	return dst;
}

__global__ void forDiffSquare(float* arr, int current, int _rows, int _columns) {
	for (int i = 0; i < _rows; ++i) {
		if (i == current) continue;
		for(int j = 0; j < _columns; ++j) {
			if (j == current) continue;
			arr[i * _columns + j] = 0;
		}
	}
}

Matrix Matrix::DiffSquare(int variable) {
	Matrix ret(*this);
	forDiffSquare << <1, 1 >> > (ret.arr, variable, _rows, _columns);
	hipDeviceSynchronize();

	return ret;
}

__global__ void increm(float* arr, int i, int j, float inc, int _rows, int _columns) {
	arr[i * _columns + j] += inc;
}

void Matrix::Increase(int i, int j, float inc) {
	increm << <1, 1 >> > (arr, i, j, inc, _rows, _columns);
	hipDeviceSynchronize();

}
__global__ void copyDiffToMatrixFunc(float* arr, float* diffArr, int variable, int _rows, int _columns, int tfr) {
	for (int j = 1; j < _columns ; ++j) {
		arr[(variable - 1) * _columns + j - 1] =
			diffArr[(variable)*_columns + j] + diffArr[(j)*_columns + variable];
	}
}

void Matrix::CopyDiffToMatrix(Matrix& diff, int variable) {
	copyDiffToMatrixFunc << <1, 1 >> > (arr, diff.arr, variable, _rows, _columns, _threadsForRow);
	hipDeviceSynchronize();
	set(variable - 1, _columns - 1, -(diff.get(variable, 0) + diff.get(0, variable)));
}

__global__ void writeToDiagFunc(float* dst, float* src, int _rows, int _columns, int tfr) {
	for (int i = 0; i < _rows; ++i) {
		dst[i * _columns + i] = src[i];
	}
}

void Matrix::WriteToDiag(float* diagArr) {
	float* src;
	hipMalloc(&src, _columns * sizeof(float));
	hipMemcpy(src, diagArr, _columns * sizeof(float), hipMemcpyHostToDevice);

	writeToDiagFunc << <1, 1 >> > (arr, src, _rows, _columns, _threadsForRow);
	hipDeviceSynchronize();;

	hipFree(src);
}